#include <hip/hip_runtime.h>

__global__ void add_kernel(const float* a, const float* b, float* c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) c[i] = a[i] + b[i];
}

extern "C" void add_launch(const float* a, const float* b, float* c, int n){
    const int bs = 256;
    const int gs = (n + bs - 1) / bs;
    add_kernel<<<gs, bs>>>(a, b, c, n);
    hipDeviceSynchronize();
}