// ...existing code...
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <vector>

extern "C" void add_launch(const float*, const float*, float*, int);

TEST(AddTest, Small){
    int n = 4;
    std::vector<float> a{1,2,3,4}, b{10,20,30,40}, c(n);
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    ASSERT_EQ(hipSuccess, hipMalloc(&d_a, n * sizeof(float)));
    ASSERT_EQ(hipSuccess, hipMalloc(&d_b, n * sizeof(float)));
    ASSERT_EQ(hipSuccess, hipMalloc(&d_c, n * sizeof(float)));

    ASSERT_EQ(hipSuccess, hipMemcpy(d_a, a.data(), n * sizeof(float), hipMemcpyHostToDevice));
    ASSERT_EQ(hipSuccess, hipMemcpy(d_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice));

    add_launch(d_a, d_b, d_c, n);

    ASSERT_EQ(hipSuccess, hipMemcpy(c.data(), d_c, n * sizeof(float), hipMemcpyDeviceToHost));

    EXPECT_FLOAT_EQ(c[0], 11.0f);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

int main(int argc,char** argv){ ::testing::InitGoogleTest(&argc,argv); return RUN_ALL_TESTS(); }
// ...existing code...